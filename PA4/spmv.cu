#include <stdio.h>
#include <stdlib.h>

extern int hipMemcpy();
extern int hipFree();

main (int argc, char **argv) 
{
  FILE *fp;
  char line[1024]; 
  int *ptr, *indices;
  float *data, *b, *t;
  int i,j;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix


// ------------------- Variables for cuda solution -----------------
  FILE* fp_c;
  int* ptr_c;
  int* indices_c;
  float* data_c;
  float* b_c;
  float* t_c;
// ------------------- end of Variables for cuda solution -----------------

  // Open input file and read to end of comments
  if (argc !=2) abort(); 

  if ((fp = fopen(argv[1], "r")) == NULL) { abort();  }

  fgets(line, 128, fp);
  while (line[0] == '%') 
  {
    fgets(line, 128, fp); 
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  //------------ cuda mallocs ------------------------
  hipMalloc(&ptr_c, (nr+1)*sizeof(int));
  hipMalloc(&indices_c, n*sizeof(int));
  hipMalloc(&data_c, n*sizeof(float));
  hipMalloc(&b_c, nc*sizeof(float));
  hipMalloc(&t_c, nr*sizeof(float));
  //------------ end of cuda mallocs -----------------

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;

  for (i=0; i<n; i++) 
  {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));

    indices[i]--;  // start numbering at 0
    
    if (r!=lastr) 
    { 
      ptr[r-1] = i;
      lastr = r; 
    }
  }

  ptr[nr] = n;

  // initialize t to 0 and b with random data  
  for (i=0; i<nr; i++) 
  {
    t[i] = 0.0;
  }

printf("segfault before?\n");
    fflush(stdout);

  for (i=0; i<nc; i++) 
  {
    b[i] = (float) rand()/1111111111;
  }

  hipMemcpy(b_c, b, nc*sizeof(float), hipMemcpyHostToDevice);

  printf("segfault after?\n");
    fflush(stdout);

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) 
  {                                                      
    for (j = ptr[i]; j<ptr[i+1]; j++) 
    {
      t[i] = t[i] + data[j] * b[indices[j]];
    }
  }

  // TODO: Compute result on GPU and compare output

//------------------------------------------------------------------------------------------
    

  if ((fp_c = fopen(argv[1], "r")) == NULL) { abort();  }

  // Read data in coordinate format and initialize sparse matrix
  int lastr_c = 0;

  for (i=0; i<n; i++) 
  {
    int r_c;
    fscanf(fp_c,"%d %d %f\n", &r_c, &(indices_c[i]), &(data_c[i]));

    indices_c[i]--;  // start numbering at 0
    
    if (r_c!=lastr_c) 
    { 
      ptr_c[r_c-1] = i;

      lastr_c = r_c; 
    }
  }

  ptr_c[nr] = n;

  // initialize t to 0 and b with random data  
  for (i=0; i<nr; i++) 
  {
    t_c[i] = 0.0;
  }

  // for (i=0; i<nc; i++) 
  // {
  //   //b_c[i] = (float) rand()/1111111111;
  //   b_c[i] = b[i];
  // }

  

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) 
  {                                                      
    for (j = ptr_c[i]; j<ptr_c[i+1]; j++) 
    {
      t[i] = t[i] + data_c[j] * b_c[indices_c[j]];
    }
  }
}
