#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

extern int hipMemcpy();
extern int hipFree();

__global__ void spmv(int nr_c, int* ptr_c, float* t_c, float* data_c, float* b_c, int* indices_c);

main (int argc, char **argv) 
{
  FILE *fp;
  char line[1024]; 
  int *ptr, *indices;
  float *data, *b, *t;
  int i,j;
  int n; // number of nonzero elements in data
  int nr; // number of rows in matrix
  int nc; // number of columns in matrix


// ------------------- Variables for cuda solution -----------------
  //FILE* fp_c;
  int* ptr_c;
  int* indices_c;
  float* data_c;
  float* b_c;
  float* t_c;
// ------------------- end of Variables for cuda solution -----------------

  // Open input file and read to end of comments
  if (argc !=2) abort(); 

  if ((fp = fopen(argv[1], "r")) == NULL) { abort();  }

  fgets(line, 128, fp);
  while (line[0] == '%') 
  {
    fgets(line, 128, fp); 
  }

  // Read number of rows (nr), number of columns (nc) and
  // number of elements and allocate memory for ptr, indices, data, b and t.
  sscanf(line,"%d %d %d\n", &nr, &nc, &n);
  ptr = (int *) malloc ((nr+1)*sizeof(int));
  indices = (int *) malloc(n*sizeof(int));
  data = (float *) malloc(n*sizeof(float));
  b = (float *) malloc(nc*sizeof(float));
  t = (float *) malloc(nr*sizeof(float));

  //------------ cuda mallocs ------------------------
  hipMalloc(&ptr_c, (nr+1)*sizeof(int));
  hipMalloc(&indices_c, n*sizeof(int));
  hipMalloc(&data_c, n*sizeof(float));
  hipMalloc(&b_c, nc*sizeof(float));
  hipMalloc(&t_c, nr*sizeof(float));
  //------------ end of cuda mallocs -----------------

  // Read data in coordinate format and initialize sparse matrix
  int lastr=0;

  for (i=0; i<n; i++) 
  {
    int r;
    fscanf(fp,"%d %d %f\n", &r, &(indices[i]), &(data[i]));

    indices[i]--;  // start numbering at 0
    
    if (r!=lastr) 
    { 
      ptr[r-1] = i;
      lastr = r; 
    }
  }

  hipMemcpy(indices_c, indices, n*sizeof(int), hipMemcpyHostToDevice);

  ptr[nr] = n;

  hipMemcpy(ptr_c, ptr, (nr+1)*sizeof(int), hipMemcpyHostToDevice);

  // initialize t to 0 and b with random data  
  for (i=0; i<nr; i++) 
  {
    t[i] = 0.0;
  }

  hipMemcpy(t_c, t, nr*sizeof(float), hipMemcpyHostToDevice);

printf("segfault before?\n");
    fflush(stdout);

  for (i=0; i<nc; i++) 
  {
    b[i] = (float) rand()/1111111111;
  }

  hipMemcpy(b_c, b, nc*sizeof(float), hipMemcpyHostToDevice);

  printf("segfault after?\n");
    fflush(stdout);

  // MAIN COMPUTATION, SEQUENTIAL VERSION
  for (i=0; i<nr; i++) 
  {                                                      
    for (j = ptr[i]; j<ptr[i+1]; j++) 
    {
      t[i] = t[i] + data[j] * b[indices[j]];
      printf("%f ", t[i]);
    }

    printf("\n");
  }

  // TODO: Compute result on GPU and compare output

  spmv(nr, ptr, t, data, b, indices);
}

void spmv(int nr_c, int* ptr_c, float* t_c, float* data_c, float* b_c, int* indices_c)
{

}

