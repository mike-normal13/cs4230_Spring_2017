#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//extern int hipMemcpy();
//extern int hipFree();

extern __global__ void cudaMatMul(int** C, int** A, int** B);

int main(int argc, char** argv)
{
	int N = 16;

	int* A[N];
	int* B[N];

	// result
	int* C[N];

	// cuda guys
	int* A_c[N];
	int* B_c[N];
	int* C_c[N];

	int i = 0;
	int j = 0;
	int k = 0;

	// malloc individual arrays
	for(i = 0; i < N; i++)
	{
		A[i] = (int*) malloc(N * sizeof(int));
		B[i] = (int*) malloc(N * sizeof(int));
		C[i] = (int*) malloc(N * sizeof(int));

		hipMalloc((void*) &A_c[i], N * sizeof(int));
		hipMalloc((void*) &B_c[i], N * sizeof(int));
		hipMalloc((void*) &C_c[i], N * sizeof(int));
	}

	// init data
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			A[i][j] = i + j;
			B[i][j] = i * j;

			C[i][j] = 0;
		}
	}

	// mat mul
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			for(k = 0; k < N; k++)
			{
				C[i][j] += A[i][k] * B[k][j];
			}
		}
	}

	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			printf("%d ", C[i][j]);
		}
		printf("\n");
	}

	// free arrays
	for(i = 0; i < N; i++)
	{
		free(A[i]);
		free(B[i]);
		free(C[i]);
	}

	return 0;
}