#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>

//extern int hipMemcpy();
//extern int hipFree();

extern __global__ 
void cudaMatMul(int** C, int** A, int** B);

int main(int argc, char** argv)
{
	int N = 16;

	int* A[N];
	int* B[N];

	// result
	int* C[N];

	// cuda guys
	int* A_c[N];
	int* B_c[N];
	int* C_c[N];

	// cuda result placed in this value
	int* ret[N];

	int i = 0;
	int j = 0;

	// malloc individual arrays
	for(i = 0; i < N; i++)
	{
		A[i] = (int*) malloc(N * sizeof(int));
		B[i] = (int*) malloc(N * sizeof(int));
		C[i] = (int*) malloc(N * sizeof(int));

		hipMalloc((void**) &A_c[i], N * sizeof(int));
		hipMalloc((void**) &B_c[i], N * sizeof(int));
		hipMalloc((void**) &C_c[i], N * sizeof(int));

		hipMalloc((void**) &ret[i], N * sizeof(int));
	}

	// init data
	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
		{
			A[i][j] = i + j;
			B[i][j] = i * j;

			C[i][j] = 0;
		}
	}

	// COPY TO device memory
	for(i = 0; i < N; i++)
	{
		hipMemcpy(A_c[i], A[i], N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(B_c[i], B[i], N * sizeof(int), hipMemcpyHostToDevice);
		hipMemcpy(C_c[i], C[i], N * sizeof(int), hipMemcpyHostToDevice);
	}

	cudaMatMul(C_c, A_c, B_c);	

	for(i = 0; i < N; i++)
	{
		hipMemcpy(ret[i], C_c[i], N * sizeof(int), hipMemcpyDeviceToHost);
	}

	for(i = 0; i < N; i++)
	{
		for(j = 0; j < N; j++)
			printf("%d ", ret[i][j]);
		printf("\n");
	}
	

	// free arrays
	for(i = 0; i < N; i++)
	{
		free(A[i]);
		free(B[i]);
		free(C[i]);

		hipFree(A_c);
		hipFree(B_c);
		hipFree(C_c);

		hipFree(ret);
	}

	return 0;
}

extern __global__ 
void cudaMatMul(int** C, int** A, int** B)
{
	int i, j, k = 0;	

	// mat mul
	for(i = 0; i < N; i++)
		for(j = 0; j < N; j++)
			for(k = 0; k < N; k++)
				C[i][j] += A[i][k] * B[k][j];
}

